#include "hip/hip_runtime.h"

#include "solver.h"
#include <assert.h>
#include <stddef.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

/** Utiles */
#define ABS(x) x < 0.0f ? -x : x

__host__ __device__
static size_t rb_idx(size_t x, size_t y, size_t dim)
{
    assert(dim % 2 == 0);
    size_t base = ((x % 2) ^ (y % 2)) * dim * (dim / 2);
    size_t offset = (y / 2) + x * (dim / 2);
    return base + offset;
}

#define IX(x, y) (rb_idx((x), (y), (N + 2)))

// Simplificar
#define SWAP(x0, x)      \
    {                    \
        float* tmp = x0; \
        x0 = x;          \
        x = tmp;         \
    }

template <typename T>
T div_ceil(T a, T b) {
    return (a + b - 1) / b;
}

/** Constantes */
#ifndef N
#define N 256
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 128
#endif

typedef enum boundary {
    NONE = 0,
    VERTICAL = 1,
    HORIZONTAL = 2 
} boundary;

__global__
void add_source(unsigned int n, float* x, const float* s, float dt) {
    uint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (n + 2) * (n + 2)) x[i] += dt * s[i];
}

static void launch_add_source(unsigned int n, float* x, const float* s, float dt) {
    dim3 block(BLOCK_SIZE);
    dim3 grid(div_ceil<uint>(n + 2 * n + 2, BLOCK_SIZE));
    add_source<<<grid,block>>>(n,x,s,dt);
    getLastCudaError("add_source() kernel failed");
	checkCudaErrors(hipDeviceSynchronize());
}

__global__
void set_bnd(unsigned int n, boundary b, float* x)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n + 1){
        x[IX(0, i)] = b == VERTICAL ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(n + 1, i)] = b == VERTICAL ? -x[IX(n, i)] : x[IX(n, i)];
        x[IX(i, 0)] = b == HORIZONTAL ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, n + 1)] = b == HORIZONTAL ? -x[IX(i, n)] : x[IX(i, n)];
    }   
}

static void launch_set_bnd(unsigned int n, boundary b, float* x){
    dim3 block(BLOCK_SIZE);
    dim3 grid(div_ceil<uint>(n, BLOCK_SIZE));
    set_bnd<<<grid,block>>>(n,b,x);
    getLastCudaError("set_bnd() kernel failed");
	checkCudaErrors(hipDeviceSynchronize());
}

static void set_corners(unsigned int n, boundary b, float* x) {
    x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(0, n + 1)] = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
    x[IX(n + 1, 0)] = 0.5f * (x[IX(n, 0)] + x[IX(n + 1, 1)]);
    x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

static void lin_solve(unsigned int n, boundary b, float* x, const float* x0, float a, float c)
{
//     int offsetI = 0, offsetF = 0, alpha = 0, base = 0;
//     float acum1, acum2, acumT ;
//     unsigned int cont1, cont2, contT;
//     unsigned int k = 0;
//     float inv_c = 1.0f / c;

//     do {
//         k++;
//         acum1 = 0.0f,acum2 = 0.0f,acumT = 0.0f;
//         cont1 = 0,cont2 = 0,contT = 0;
        
//         // Impar - Impar
//         #pragma omp parallel shared(x,x0, a, b, inv_c, n, contT, acumT) private(base, offsetI, offsetF, alpha) 
//         {
//         base = (n * n / 2) + 1;
//         offsetI = 0;
//         offsetF = -1;
//         alpha = -1;
//         #pragma omp for reduction(+:cont1, acum1)
//         for (size_t i = 1; i < n - 1; i += 2)
//             lin_solve_single(n+2, i, base, offsetI, offsetF, &cont1, &acum1, alpha, x, x0, a, inv_c);
        
//         /// Rojos ; Par - Par
//         base = (n * n / 2) - 1;
//         offsetI = 1;
//         offsetF = 0;
//         alpha = 1;

//         #pragma omp for reduction(+:cont2, acum2)
//         for (size_t i = 2; i < n - 1; i += 2)
//             lin_solve_single(n+2, i, base, offsetI, offsetF, &cont2, &acum2, alpha, x, x0, a, inv_c);
        
//         #pragma omp barrier
//         acumT += acum1 + acum2;
//         contT += cont1 + cont2;

//         cont1 = 0,cont2 = 0; 
//         acum1 = 0.0f,acum2 = 0.0f;        

//         /// Negros ; Par - Impar
//         offsetI = n * n / 2;
//         offsetF = n * n / 2 - 1;
//         base = -((n * n / 2) - 1);
//         alpha = -1;

//         #pragma omp for reduction(+:cont1, acum1)
//         for (size_t i = 1; i < n - 1; i += 2)
//             lin_solve_single(n+2, i, base, offsetI, offsetF, &cont1, &acum1, alpha, x, x0, a, inv_c);

//         /// Negros ; Impar - Par
//         base = -((n * n / 2) + 1);
//         offsetI = n * n / 2 + 1;
//         offsetF = n * n / 2;
//         alpha = 1;

//         #pragma omp for reduction(+:cont2, acum2)
//         for (size_t i = 2; i < n - 1; i += 2)
//             lin_solve_single(n+2, i, base, offsetI, offsetF, &cont2, &acum2, alpha, x, x0, a, inv_c);

//         #pragma omp barrier
//         acumT += acum1 + acum2;
//         contT += cont1 + cont2;
//         }
//         set_bnd(n, b, x);s
        
//     } while (acumT / (float) contT > 1e-10f && k < 20);

// #else
    for (unsigned int k = 0; k < 20; k++) {
        for (unsigned int i = 1; i < n + 1; i++) {
            for (unsigned int j = 1; j < n + 1; j++) {
                x[IX(i, j)] = (x0[IX(i, j)]
                               + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)]))
                    / c;
            }
        }
        launch_set_bnd(n, b, x);
        set_corners(n, b, x);
    }
}

static void launch_lin_solve(unsigned int n, boundary b, float* x, const float* x0, float a, float c) {
    // dim3 block(1,1);
    // dim3 grid(1,1);
    // pre_project<<<grid, block>>>(n, u, v, p, div);
    // getLastCudaError("pre_project() kernel failed");
    // checkCudaErrors(hipDeviceSynchronize());
}


static void diffuse(unsigned int n, boundary b, float* x, const float* x0, float diff, float dt)
{
    float a = dt * diff * n * n;
    launch_lin_solve(n, b, x, x0, a, 1 + 4 * a);
}

__global__
void advect_step(unsigned int n, boundary b, float* d, const float* d0, const float* u, const float* v, float dt) {
    int i0, i1, j0, j1;
    float x, y, s0, t0, s1, t1;

    float dt0 = dt * n;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < n+1 && j < n+1) {
        x = i - dt0 * u[IX(i, j)];
        y = j - dt0 * v[IX(i, j)];
        if (x < 0.5f)
            x = 0.5f;
        else if (x > n + 0.5f)
            x = n + 0.5f;
        i0 = (int)x;
        i1 = i0 + 1;
        if (y < 0.5f)
            y = 0.5f;
        else if (y > n + 0.5f)
            y = n + 0.5f;
        j0 = (int)y;
        j1 = j0 + 1;
        s1 = x - i0;
        s0 = 1 - s1;
        t1 = y - j0;
        t0 = 1 - t1;
        d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) + s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    }
}

static void launch_advect_step(unsigned int n, boundary b, float* d, const float* d0, const float* u, const float* v, float dt) {
    dim3 block(1,1);
    dim3 grid(1,1);
    advect_step<<<grid, block>>>(n, b, d, d0, u, v, dt);
    getLastCudaError("advect_step() kernel failed");
    checkCudaErrors(hipDeviceSynchronize());
}

static void advect(unsigned int n, boundary b, float* d, const float* d0, const float* u, const float* v, float dt)
{
    launch_advect_step(n,b,d,d0,u,v,dt);
    launch_set_bnd(n, b, d);
    set_corners(n, b, d);
}

__global__
void pre_project(unsigned int n, float* u, float* v, float* p, float* div){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < n + 1 && j < n + 1) {
        div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
        p[IX(i, j)] = 0;
    }    
}

static void launch_pre_project(unsigned int n, float* u, float* v, float* p, float* div) {
    dim3 block(1,1);
    dim3 grid(1,1);
    pre_project<<<grid, block>>>(n, u, v, p, div);
    getLastCudaError("pre_project() kernel failed");
    checkCudaErrors(hipDeviceSynchronize());
}

__global__
void post_project(unsigned int n, float* u, float* v, float* p, float* div){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < n + 1 && j < n + 1) {
        u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
        v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
    }
}

static void launch_post_project(unsigned int n, float* u, float* v, float* p, float* div) {
    dim3 block(1,1);
    dim3 grid(1,1);
    pre_project<<<grid, block>>>(n, u, v, p, div);
    getLastCudaError("post_project() kernel failed");
    checkCudaErrors(hipDeviceSynchronize());
}

static void project(unsigned int n, float* u, float* v, float* p, float* div)
{
    launch_pre_project(n,u,v,p,div);

    launch_set_bnd(n, NONE, div);
    set_corners(n, NONE, div);

    launch_set_bnd(n, NONE, p);
    set_corners(n, NONE, p);

    launch_lin_solve(n, NONE, p, div, 1, 4);

    launch_post_project(n,u,v,p,div);
    
    launch_set_bnd(n, VERTICAL, u);
    set_corners(n, VERTICAL, u);

    launch_set_bnd(n, HORIZONTAL, v);
    set_corners(n, HORIZONTAL, v);
}

void dens_step(unsigned int n, float* x, float* x0, float* u, float* v, float diff, float dt)
{
    launch_add_source(n, x, x0, dt);
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

void vel_step(unsigned int n, float* u, float* v, float* u0, float* v0, float visc, float dt)
{
    launch_add_source(n, u, u0, dt);
    launch_add_source(n, v, v0, dt);
    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
