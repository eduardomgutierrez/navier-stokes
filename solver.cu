#include "hip/hip_runtime.h"
#include "solver.h"
#include <assert.h>
#include <stddef.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

/** Constantes */
#ifndef N
#define N 256
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 128
#endif

#ifndef BLOCK_SIZE_2D
#define BLOCK_SIZE_2D 32
#endif
typedef enum boundary { NONE = 0, VERTICAL = 1, HORIZONTAL = 2 } boundary;


/** Utiles */
#define ABS(x) x < 0.0f ? -x : x

#define IX(i, j) ((i) + (n + 2) * (j))
// __device__ size_t IX(size_t x, size_t y)
// {
//     size_t dim = N + 2;
//     assert(dim % 2 == 0);
//     size_t base = ((x % 2) ^ (y % 2)) * dim * (dim / 2);
//     size_t offset = (y / 2) + x * (dim / 2);
//     return base + offset;
// }

// Simplificar
#define SWAP(x0, x)      \
    {                    \
        float* tmp = x0; \
        x0 = x;          \
        x = tmp;         \
    }
    
template <typename T>
T div_ceil(T a, T b) {
    return (a + b - 1) / b;
}

__global__ void add_source(uint n, float* x, const float* s, float dt) {
    uint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (n + 2) * (n + 2))
        x[i] += dt * s[i];
}

static void launch_add_source(uint n, float* x, const float* s, float dt) {
    dim3 block(BLOCK_SIZE);
    dim3 grid(div_ceil<uint>((n + 2) * (n + 2), BLOCK_SIZE));
    add_source<<<grid,block>>>(n, x, s, dt);
    getLastCudaError("add_source() kernel failed");
}

__global__
void set_bnd(uint n, boundary b, float* x)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    if (i < n + 1){
        x[IX(0, i)] = b == VERTICAL ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(n + 1, i)] = b == VERTICAL ? -x[IX(n, i)] : x[IX(n, i)];
        x[IX(i, 0)] = b == HORIZONTAL ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, n + 1)] = b == HORIZONTAL ? -x[IX(i, n)] : x[IX(i, n)];
    }
    __syncthreads();
}

static void launch_set_bnd(uint n, boundary b, float* x){
    dim3 block(BLOCK_SIZE);
    dim3 grid(div_ceil<uint>(n, BLOCK_SIZE));
    set_bnd<<<grid,block>>>(n,b,x);
    getLastCudaError("set_bnd() kernel failed");
	checkCudaErrors(hipDeviceSynchronize());
}

__global__
void set_corners(uint n, boundary b, float* x) {
    x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(0, n + 1)] = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
    x[IX(n + 1, 0)] = 0.5f * (x[IX(n, 0)] + x[IX(n + 1, 1)]);
    x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

static void launch_set_corners(uint n, boundary b, float* x) {
    set_corners<<<1,1>>>(n,b,x);
    getLastCudaError("set_bnd() kernel failed");
	checkCudaErrors(hipDeviceSynchronize());
}

__global__
void lin_solve_step(uint n,
                    uint k, 
                    uint base, 
                    uint offsetI, 
                    uint offsetF, 
                    uint * cont, 
                    float * acum, 
                    uint alpha,
                    float *x, 
                    const float *x0, 
                    float a, 
                    float inv_c)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int j = blockDim.y * blockIdx.y + threadIdx.y + 1;
    
    x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) * inv_c;

// foreach(idx = offsetI + i * n/2 ...  offsetF +(i+1) * n/2) 
//     {
//         x[idx] = (x0[idx]
//             + a * (x[idx - (n/2 - alpha) + base]
//             + x[idx + (n/2 + alpha) + base]
//             + x[idx + base + alpha]
//             + x[idx + base])) * inv_c;
//         if(abs(x[idx]) > 1e-10f){
//             cont_ ++;         
//             acum_ += abs(x[idx]-x0[idx]);
//         }
//     }
}
    

static void launch_lin_solve_step(uint n, 
                                  uint i,
                                  uint base,
                                  uint offsetI,
                                  uint offsetF,
                                  uint * cont,
                                  float * acum,
                                  uint alpha, 
                                  float *x,
                                  const float *x0,
                                  float a,
                                  float inv_c)
{
    dim3 block(BLOCK_SIZE_2D,BLOCK_SIZE_2D);
    dim3 grid(div_ceil(n, block.x), div_ceil(n, block.y));
    lin_solve_step<<<grid, block>>>(n, i, base, offsetI, offsetF, cont, acum, alpha, x, x0, a, inv_c);
    getLastCudaError("lin_solve_step() kernel failed");
}


static void lin_solve(uint n, boundary b, float* x, const float* x0, float a, float c)
{
    int offsetI = 0, offsetF = 0, alpha = 0, base = 0;
    // float acum1, acum2, acumT ;
    // uint cont1, cont2, contT;
    uint k = 0;
    float inv_c = 1.0f / c;

    do {
        k++;
        // acum1 = 0.0f,acum2 = 0.0f,acumT = 0.0f; // cont1 = 0,cont2 = 0,contT = 0;
        
        // // Impar - Impar
        // base = (n * n / 2) + 1;
        // offsetI = 0;
        // offsetF = -1;
        // alpha = -1;
        // for (size_t i = 1; i < n - 1; i += 2)
        //     launch_lin_solve_step(n, i, base, offsetI, offsetF, nullptr, nullptr, alpha, x, x0, a, inv_c);

        // /// Rojos ; Par - Par
        // base = (n * n / 2) - 1;
        // offsetI = 1;
        // offsetF = 0;
        // alpha = 1;
        // for (size_t i = 2; i < n - 1; i += 2)
        //     launch_lin_solve_step(n, i, base, offsetI, offsetF, nullptr, nullptr, alpha, x, x0, a, inv_c);
        
        // checkCudaErrors(hipDeviceSynchronize());
        // // acumT += acum1 + acum2; // contT += cont1 + cont2; // cont1 = 0,cont2 = 0;  // acum1 = 0.0f,acum2 = 0.0f;        

        // /// Negros ; Par - Impar
        // offsetI = n * n / 2;
        // offsetF = n * n / 2 - 1;
        // base = -((n * n / 2) - 1);
        // alpha = -1;

        // #pragma omp for reduction(+:cont1, acum1)
        // for (size_t i = 1; i < n - 1; i += 2)
        //     launch_lin_solve_step(n, i, base, offsetI, offsetF, nullptr, nullptr, alpha, x, x0, a, inv_c);

        // /// Negros ; Impar - Par
        // base = -((n * n / 2) + 1);
        // offsetI = n * n / 2 + 1;
        // offsetF = n * n / 2;
        // alpha = 1;

        // for (size_t i = 2; i < n - 1; i += 2)
        //     launch_lin_solve_step(n, i, base, offsetI, offsetF, nullptr, nullptr, alpha, x, x0, a, inv_c);

        // checkCudaErrors(hipDeviceSynchronize());

        // // acumT += acum1 + acum2; // contT += cont1 + cont2;

        launch_lin_solve_step(n, 0, base, offsetI, offsetF, nullptr, nullptr, alpha, x, x0, a, inv_c);
        checkCudaErrors(hipDeviceSynchronize());

        launch_set_bnd(n, b, x);
        launch_set_corners(n,b,x);
    
    } while (k < 20);
}

static void diffuse(uint n, boundary b, float* x, const float* x0, float diff, float dt)
{
    float a = dt * diff * n * n;
    lin_solve(n, b, x, x0, a, 1 + 4 * a);
}

__global__
void advect_step(uint n, boundary b, float* d, const float* d0, const float* u, const float* v, float dt) {
    int i0, i1, j0, j1;
    float x, y, s0, t0, s1, t1;

    float dt0 = dt * n;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < n+1 && j < n+1) {
        x = i - dt0 * u[IX(i, j)];
        y = j - dt0 * v[IX(i, j)];
        if (x < 0.5f)
            x = 0.5f;
        else if (x > n + 0.5f)
            x = n + 0.5f;
        i0 = (int)x;
        i1 = i0 + 1;
        if (y < 0.5f)
            y = 0.5f;
        else if (y > n + 0.5f)
            y = n + 0.5f;
        j0 = (int)y;
        j1 = j0 + 1;
        s1 = x - i0;
        s0 = 1 - s1;
        t1 = y - j0;
        t0 = 1 - t1;
        d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) + s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    }
}

static void launch_advect_step(uint n, boundary b, float* d, const float* d0, const float* u, const float* v, float dt) {
    dim3 block(BLOCK_SIZE_2D,BLOCK_SIZE_2D);
    dim3 grid(div_ceil(n, block.x), div_ceil(n, block.y));
    advect_step<<<grid, block>>>(n, b, d, d0, u, v, dt);
    getLastCudaError("advect_step() kernel failed");
    checkCudaErrors(hipDeviceSynchronize());
}

static void advect(uint n, boundary b, float* d, const float* d0, const float* u, const float* v, float dt)
{
    launch_advect_step(n,b,d,d0,u,v,dt);
    launch_set_bnd(n, b, d);
    launch_set_corners(n, b, d);
}

__global__
void pre_project(uint n, float* u, float* v, float* p, float* div){
    uint i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    uint j = blockDim.y * blockIdx.y + threadIdx.y + 1;
        
    if (i < n + 1 && j < n + 1) {
        div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
        p[IX(i, j)] = 0;
    }
}

static void launch_pre_project(uint n, float* u, float* v, float* p, float* div) {
    dim3 block(BLOCK_SIZE_2D,BLOCK_SIZE_2D);
    dim3 grid(div_ceil(n, block.x), div_ceil(n, block.y));
    pre_project<<<grid, block>>>(n, u, v, p, div);
    getLastCudaError("pre_project() kernel failed");
}

__global__
void post_project(uint n, float* u, float* v, float* p, float* div){
    int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int j = blockDim.y * blockIdx.y + threadIdx.y + 1;
    if (i < n + 1 && j < n + 1) {
        u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
        v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
    }
}

static void launch_post_project(uint n, float* u, float* v, float* p, float* div) {
    dim3 block(BLOCK_SIZE_2D,BLOCK_SIZE_2D);
    dim3 grid(div_ceil(n, block.x), div_ceil(n, block.y));
    post_project<<<grid, block>>>(n, u, v, p, div);
    getLastCudaError("post_project() kernel failed");
}

static void project(uint n, float* u, float* v, float* p, float* div)
{
    launch_pre_project(n,u,v,p,div);
    checkCudaErrors(hipDeviceSynchronize());

    launch_set_bnd(n, NONE, div);
    launch_set_corners(n, NONE, div);

    launch_set_bnd(n, NONE, p);
    launch_set_corners(n, NONE, p);

    lin_solve(n, NONE, p, div, 1, 4);

    launch_post_project(n,u,v,p,div);
    checkCudaErrors(hipDeviceSynchronize());
    
    launch_set_bnd(n, VERTICAL, u);
    launch_set_corners(n, VERTICAL, u);

    launch_set_bnd(n, HORIZONTAL, v);
    launch_set_corners(n, HORIZONTAL, v);
}

void dens_step(uint n, float* x, float* x0, float* u, float* v, float diff, float dt)
{
    launch_add_source(n, x, x0, dt);
    checkCudaErrors(hipDeviceSynchronize());
    
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

void vel_step(uint n, float* u, float* v, float* u0, float* v0, float visc, float dt)
{
    launch_add_source(n, u, u0, dt);
    launch_add_source(n, v, v0, dt);
    checkCudaErrors(hipDeviceSynchronize());

    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
