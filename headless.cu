/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

	This code is a simple prototype that demonstrates how to use the
	code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
	for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include "wtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "solver.h"

/* global variables */
#ifndef N
#define N 256
#endif

__host__ __device__
static size_t IX(size_t x, size_t y)
{
    size_t dim = N + 2;
    assert(dim % 2 == 0);
    size_t base = ((x % 2) ^ (y % 2)) * dim * (dim / 2);
    size_t offset = (y / 2) + x * (dim / 2);
    return base + offset;
}

#ifndef Ntimes
#define Ntimes 2048
#endif

static float dt, diff, visc;
static float force, source;

static float *u, *u_prev;
static float *v, *v_prev;
static float *dens, *dens_prev;

static void free_data(void)
{
    
    if (u)    
        checkCudaErrors(hipFree(u)); 
    if (v)
        checkCudaErrors(hipFree(v)); 
    if (u_prev)
        checkCudaErrors(hipFree(u_prev));
    if (v_prev)
        checkCudaErrors(hipFree(v_prev));
    if (dens)
        checkCudaErrors(hipFree(dens)); 
    if (dens_prev)
        checkCudaErrors(hipFree(dens_prev));
}

// Allocate and clean! 
static int allocate_data(void)
{
    int size = (N + 2) * (N + 2);
    /* Allocate magic mem in CPU & GPU. */ 
    checkCudaErrors(hipMallocManaged(&u,         size * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&v,         size * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&u_prev,    size * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&v_prev,    size * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&dens,      size * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&dens_prev, size * sizeof(float)));

    checkCudaErrors(hipMemset(u,         0, size * sizeof(float)));
    checkCudaErrors(hipMemset(v,         0, size * sizeof(float)));
    checkCudaErrors(hipMemset(u_prev,    0, size * sizeof(float)));
    checkCudaErrors(hipMemset(v_prev,    0, size * sizeof(float)));
    checkCudaErrors(hipMemset(dens,      0, size * sizeof(float)));
    checkCudaErrors(hipMemset(dens_prev, 0, size * sizeof(float)));

    if (!u || !v || !u_prev || !v_prev || !dens || !dens_prev) {
        fprintf(stderr, "cannot allocate data\n");
        return (0);
    }
    return (1);
}

static void react(float* d, float* u, float* v)
{
    int i, size = (N + 2) * (N + 2);
    float max_velocity2 = 0.0f;
    float max_density = 0.0f;

    max_velocity2 = max_density = 0.0f;
    for (i = 0; i < size; i++) {
        if (max_velocity2 < u[i] * u[i] + v[i] * v[i]) {
            max_velocity2 = u[i] * u[i] + v[i] * v[i];
        }
        if (max_density < d[i]) {
            max_density = d[i];
        }
    }

    for (i = 0; i < size; i++) {
        u[i] = v[i] = d[i] = 0.0f;
    }

    if (max_velocity2 < 0.0000005f) {
        u[IX(N / 2, N / 2)] = force * 10.0f;
        v[IX(N / 2, N / 2)] = force * 10.0f;
    }

    if (max_density < 1.0f) {
        d[IX(N / 2, N / 2)] = source * 10.0f;
    }

    return;
}

static void one_step(double* rct, double* vel, double* dns)
{
    float start_t = 0.0;

    start_t = wtime();
    react(dens_prev, u_prev, v_prev);
    *rct += (wtime() - start_t);

    start_t = wtime();
    vel_step(N, u, v, u_prev, v_prev, visc, dt);
    *vel += (wtime() - start_t);

    start_t = wtime();
    dens_step(N, dens, dens_prev, u, v, diff, dt);
    *dns += (wtime() - start_t);
}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main(int argc, char** argv)
{
    int i = 0;
    if (argc != 1 && argc != 8) {
        fprintf(stderr, "usage : %s N dt diff visc force source\n", argv[0]);
        fprintf(stderr, "where:\n");
        fprintf(stderr, "\t N      : grid resolution\n");
        fprintf(stderr, "\t dt     : time step\n");
        fprintf(stderr, "\t diff   : diffusion rate of the density\n");
        fprintf(stderr, "\t visc   : viscosity of the fluid\n");
        fprintf(stderr, "\t force  : scales the mouse movement that generate a force\n");
        fprintf(stderr, "\t source : amount of density that will be deposited\n");
        fprintf(stderr, "\t file   : output file name\n");
        exit(1);
    }

    if (argc == 1) {
        dt = 0.1f;
        diff = 0.0f;
        visc = 0.0f;
        force = 5.0f;
        source = 100.0f;
        fprintf(stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force=%g source=%g\n",
                N, dt, diff, visc, force, source);
    } else {
        dt = atof(argv[2]);
        diff = atof(argv[3]);
        visc = atof(argv[4]);
        force = atof(argv[5]);
        source = atof(argv[6]);
    }

    if (!allocate_data()) {
        exit(1);
    }

    double rct, vel, dns;

    for (i = 0; i < Ntimes; i++)
        one_step(&rct, &vel, &dns);

    long long unsigned int total = (long long unsigned int)N * (long long unsigned int)N * (long long unsigned int)Ntimes;
    printf("# CELL_MS: %f\n", (total / (rct + vel + dns)) * 1e-3);

    free_data();

    exit(0);
}
