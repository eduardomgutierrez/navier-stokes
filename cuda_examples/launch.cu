#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <hip/hip_runtime.h>
#include <cstdint>

#define WIDTH 1200
#define HEIGHT 600

struct RGBA {
    uint8_t r;
    uint8_t g;
    uint8_t b;
    uint8_t a;
};

template <typename T>
T div_ceil(T a, T b) {
    return (a + b - 1) / b;
}


// Un hilo por elemento sobre los W*H elementos de la matriz vistos en 1D
__global__ void kernel_1d(int width, int height, RGBA *rgba) {
    uint idx = blockDim.x * blockIdx.x + threadIdx.x;

    uint8_t blockColor = blockIdx.x * 255 / gridDim.x;
    uint8_t threadColor = threadIdx.x * 255 / blockDim.x;

    if (idx < width * height) {
        RGBA pixel = {blockColor, threadColor, 0u, 255u};
        rgba[idx] = pixel;
    }
}

static void launch_kernel_1d(int width, int height, RGBA *rgba, int block_size) {
    int n = width * height;
    dim3 block(block_size);
    dim3 grid(div_ceil(n, block_size));

    kernel_1d<<<grid, block>>>(width, height, rgba);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());
}


// 1D como el primero, pero cada hilo procesa N elementos consecutivos
__global__ void kernel_1d_n_items_seq(int width, int height, RGBA *rgba, int work) {

    uint8_t blockColor = blockIdx.x * 255 / gridDim.x;
    uint8_t threadColor = threadIdx.x * 255 / blockDim.x;

    int thread_from = work * (blockDim.x * blockIdx.x + threadIdx.x);
    int thread_to = min(work * (blockDim.x * blockIdx.x + threadIdx.x + 1), width * height);
    for (int idx = thread_from;
         idx < thread_to;
         ++idx)
    {
        RGBA pixel = {blockColor, threadColor, 0, 255u};
        rgba[idx] = pixel;
    }
}

static void launch_kernel_1d_n_items_seq(int width, int height, RGBA *rgba, int block_size, int work) {
    int n = width * height;
    dim3 block(block_size);
    dim3 grid(div_ceil(n, block_size * work));

    kernel_1d_n_items_seq<<<grid, block>>>(width, height, rgba, work);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());
}


// 1D como el primero, pero cada hilo procesa N elementos separados por el tamaño del bloque
__global__ void kernel_1d_n_items_block_stride(int width, int height, RGBA *rgba, int work) {

    uint8_t blockColor = blockIdx.x * 255 / gridDim.x;
    uint8_t threadColor = threadIdx.x * 255 / blockDim.x;

    int block_from = work * blockDim.x * blockIdx.x;
    int block_to = min(work * blockDim.x * (blockIdx.x + 1), width * height);
    for (int idx = block_from + threadIdx.x;
         idx < block_to;
         idx += blockDim.x)
    {
        RGBA pixel = {blockColor, threadColor, 0, 255u};
        rgba[idx] = pixel;
    }
}

static void launch_kernel_1d_n_items_block_stride(int width, int height, RGBA *rgba, int block_size, int work) {
    int n = width * height;
    dim3 block(block_size);
    dim3 grid(div_ceil(n, block_size * work));

    kernel_1d_n_items_block_stride<<<grid, block>>>(width, height, rgba, work);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());
}


// 1d como el primero, pero lanzamos una cantidad arbitraria de bloques y el grid completo avanza hasta terminar el arreglo
__global__ void kernel_1d_grid_stride(int width, int height, RGBA *rgba) {
    for (int idx = blockDim.x * blockIdx.x + threadIdx.x;
         idx < width * height;
         idx += gridDim.x * blockDim.x)
    {
        uint8_t blockColor = blockIdx.x * 255 / gridDim.x;
        uint8_t threadColor = threadIdx.x * 255 / blockDim.x;

        RGBA pixel = {blockColor, threadColor, 0, 255u};
        rgba[idx] = pixel;
    }
}

static void launch_kernel_1d_grid_stride(int width, int height, RGBA *rgba, int block_size, int num_blocks) {
    dim3 block(block_size);
    dim3 grid(num_blocks);

    kernel_1d_grid_stride<<<grid, block>>>(width, height, rgba);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());
}


// Bloques 2D dispuestos sobre la matriz vista en 2D, un elemento por hilo
__global__ void kernel_2d(int width, int height, RGBA *rgba) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    uint8_t blockColorX = blockIdx.x * 255 / gridDim.x;
    uint8_t blockColorY = blockIdx.y * 255 / gridDim.y;

    int local_id = threadIdx.x + threadIdx.y * blockDim.x;
    uint8_t threadColor = local_id * 255 / (blockDim.x * blockDim.y);

    if ((x < width) && (y < height)) {
        RGBA pixel = {blockColorX, threadColor, blockColorY, 255u};
        rgba[y * WIDTH + x] = pixel;
    }
}

static void launch_kernel_2d(int width, int height, RGBA *rgba, int block_width, int block_height) {
    dim3 block(block_width, block_height);
    dim3 grid(div_ceil(width, block_width), div_ceil(height, block_height));

    kernel_2d<<<grid, block>>>(width, height, rgba);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());
}


int main() {
    RGBA *img;
    checkCudaCall(hipMallocManaged(&img, WIDTH * HEIGHT * sizeof(RGBA)));

    checkCudaCall(hipMemset(img, 0, WIDTH * HEIGHT * sizeof(RGBA)));
    launch_kernel_1d(WIDTH, HEIGHT, img, 1024);
    stbi_write_png("1d.png", WIDTH, HEIGHT, 4, img, sizeof(RGBA) * WIDTH);

    checkCudaCall(hipMemset(img, 0, WIDTH * HEIGHT * sizeof(RGBA)));
    launch_kernel_1d_n_items_seq(WIDTH, HEIGHT, img, 1024, 4);
    stbi_write_png("1d_n_items_seq.png", WIDTH, HEIGHT, 4, img, sizeof(RGBA) * WIDTH);

    checkCudaCall(hipMemset(img, 0, WIDTH * HEIGHT * sizeof(RGBA)));
    launch_kernel_1d_n_items_block_stride(WIDTH, HEIGHT, img, 1024, 4);
    stbi_write_png("1d_n_items_block.png", WIDTH, HEIGHT, 4, img, sizeof(RGBA) * WIDTH);

    checkCudaCall(hipMemset(img, 0, WIDTH * HEIGHT * sizeof(RGBA)));
    launch_kernel_1d_grid_stride(WIDTH, HEIGHT, img, 1024, 200);
    stbi_write_png("1d_grid_stride.png", WIDTH, HEIGHT, 4, img, sizeof(RGBA) * WIDTH);

    checkCudaCall(hipMemset(img, 0, WIDTH * HEIGHT * sizeof(RGBA)));
    launch_kernel_2d(WIDTH, HEIGHT, img, 32, 32);
    stbi_write_png("2d.png", WIDTH, HEIGHT, 4, img, sizeof(RGBA) * WIDTH);

    checkCudaCall(hipFree(img));

    return 0;
}
